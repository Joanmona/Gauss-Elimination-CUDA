#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h> 


////CUDA uses a C++ compiler to compile .cu files. Therefore, it expects that all functions referenced in .cu files
////have C++ linkage unless explicitly instructed otherwise. 
////And in my case, I must explicitly instruct the C++ compiler otherwise.

extern "C" { 
#include "Read_And_Write_Linear.h"
}


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>

#include "cuPrintf.cu"

//για τον έλεγχο των σφαλμάτων
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
} 

/////////////////////
/////// Parallel In GPU
////////////////

int *in_data;           //store array in CPU
int *in_gpu_data;  //store array in GPU

int *lines_one, *gpu_lines_one; // Χρήση για την εύρεση της γραμμής που θα χρησιμοποιθεί στην xor
int *lines_zeros, *gpu_lines_zeros; // Χρήση για το μέτρημα των μηδενικών

clock_t start, end;  // Μετράνε το χρόνο εκτέλεσης του προγράμματος
double cpu_time_used; // Ο τελικός χρόνος εκτέλεσης του προγράμματος




/*
 * Η Συνάρτηση αυτή εκτελείται από τη GPU και κάνει swap τις γραμμές που έχουμε επιλέξει.
 * Δέχεται σαν παραμέτρους τον πίνακα μετά από κάθε "κύκλο" του Gauss, τον αριθμό των στηλών, των γραμμών,τη θέση pivot (at),
 * τη στήλη στην οποία βρισκόμαστε, τη γραμμή στην οποία βρισκόμαστε και τη γραμμή με την οποία θα γίνει 
 * η ανταλλαγή (swap_line). 
 * Κάθε thread είναι υπεύθυνο να κάνει swap τη θέση της γραμμής του pivot με την αντίστοιχη θέση στη swap_line και είναι στην
 * ίδια στήλη για την οποία είναι υπεύθυνο. 
 * 
 */

__global__ void swap(int *in_gpu_data, int cols, int lines, int at, int start_col, int start_line, int swap_line)
{

		int i = blockDim.x * blockIdx.x + threadIdx.x;  //global id
		//printf(" i = %d \n",i);
		//printf(" blockDim.x = %d \n",blockDim.x);
		//printf(" blockIdx.x = %d \n",blockIdx.x );
		//printf(" threadIdx.x = %d \n",threadIdx.x);
		
       if (i==1)
			{
				cuPrintf ( " SWAP \n ");
			}
   
    
		__syncthreads(); // Για να ξυπνήσουν όλα τα threads 

		
		      
    	// τα αχρηστα/περισευούμενα τα threads να μη δουλευουν
	    // ο αριθμός των threads που θα χρειαστούν είναι ίσο με τον αριθμό των στηλών που θα γίνει ανταλλαγή
		//Δηλαδή από το pivot και μετά. 
				  
                      
		if (i<(cols-start_col))
			{
					   
				//cuPrintf(" iii = %d \n", i);
				
			    //Κάθε thread είναι υπεύθυνο για την ανταλλαγή των 2 θέσεων που βρίσκονται στην ίδια στήλη.
                       
					        
			    int h = swap_line-start_line;
			    
			     //at είναι η θεση στην οποία βρισκόμαστε, και οι υπόλοιπες θέσεις που θα ανταλλαχτούν (a1) από 
			     //τη γραμμή στην οποία βρισκόμαστε (η γραμμή που έχει pivot=0) βρίσκονται πάνω στην ίδια γραμμή
			     //αλλά σε διαφορετικές στήλες. Για το at είναι υπεύθυνο το thread 0, για την επόμενη θεση (at+1)
			     //είναι υπεύθυνο το thread 1 κτλ.
			    int a1 = at + i;
               
                  // Το a2 είναι οι θέσεις της γραμμής με την οποία θα ανταλλαχτεί η πάνω γραμμή. Η κάθε θέση
                  //βρίσκεται στην ίδια στήλη με τη θέση που θα κάνει ανταλλαγή αλλά μερικές γραμμές κάτω.
                 //Το h καθορίζει πόσες γραμμές κάτω είναι, η διαφορά τους σε μία γραμμή
                 //είναι όσο ο αριθμός των στηλών.
               int a2 = at + cols*h + i;
                            
			   //cuPrintf(" a1 i= %d %d\n", a1,i);
			   //cuPrintf(" a2 i= %d %d\n", a2,i);
                     
               //Κάνε ανταλλαγή τις θέσεις
			   int tt = in_gpu_data[a1];
			   in_gpu_data[a1] = in_gpu_data[a2];
			   in_gpu_data[a2] = tt;

		    }
   
		__syncthreads();
				  

}



/*
 * Η Συνάρτηση αυτή εκτελείται από τη GPU και κάνει xor τις γραμμές που έχουν 1 στην ίδια στήλη που βρίσκεται το pivot.
 * Δέχεται σαν παραμέτρους τον πίνακα μετά από κάθε "κύκλο" του Gauss, τον αριθμό των στηλών, των γραμμών,τη θέση pivot (at),
 * τη στήλη στην οποία βρισκόμαστε, τη γραμμή στην οποία βρισκόμαστε και τον πίνακα gpu_lines_ones που περιέχει τον αριθμό 1
 * στις θέσεις όπου οι αντίστοιχες γραμμές έχουν 1 στην ίδια στήλη με το pivot. 
 * Κάθε thread είναι υπεύθυνο να κάνει xor τη θέση της γραμμής του pivot με τις θέσεις που ανήκουν στη στήλη 
 * για την οποία είναι υπεύθυνα για όλες τις γραμμές που έχουν 1 στον πίνακα gpu_lines_one. 
 * 
 */

__global__ void xor_(int *in_gpu_data, int cols, int lines,int at, int start_col, int start_line, int *gpu_lines_one)
{

         int i = blockDim.x * blockIdx.x + threadIdx.x; 
	    //printf(" i = %d \n",i);
		//printf(" blockDim.x = %d \n",blockDim.x);
		//printf(" blockIdx.x = %d \n",blockIdx.x );
		//printf(" threadIdx.x = %d \n",threadIdx.x);
     
        if (i==1)
          {
           cuPrintf ( " XOR \n ");
          }
    
	    
		   __syncthreads(); // Για να ξυπνήσουν όλα τα threads 
		
		
			  
		int perisema = cols-start_col;
			  
		//τα threads που χρειάζονται είναι όσα είναι οι στήλες που θα κάνουν xor, δηλαδή από τη στήλη του pivot 
		//(μαζί με αυτή του pivot) και πέρα
		
	    if( i>=0 && i<perisema) //Θέλω να κάνω xor και τη στήλη στην οποία είμαι (start_col)
			  	 {
	    	
				  // cuPrintf("perisema i = %d %d\n", perisema,i);
				  //cuPrintf("pcols i = %d %d\n", cols,i);
				  //cuPrintf("pstart i = %d %d\n", start_col,i);
	    	
	    	
				  int at1 = at + i;// Οι θέσεις που θα γίνουν xor αλλά δε θα αλλάξουν κινούνται πάνω στην ίδια γραμμή 
				  //cuPrintf("at1 i = %d %d\n",at1,i);
				  
				  for ( int f=0; f < lines ; f++ ) // για κάθε γραμμή, βρες ποιες γραμμές έχουν 1 κάτω από το pivot
				  	  {
				          
					     if (gpu_lines_one[f] == 1) // αν η θέση έχει 1 τότε κάνε xor με την αντίστοιχη γραμμή
					     {
					    	 int grammh = f;
					    	 int jumps = grammh - start_line; // πόσες γραμμές θα πρέπει να κατέβει για να πάει στη γραμμή με την οποία θα κάνει xor
					    	 int at2 = at1 + cols*jumps; //θέση με την οποία θα κάνει xor. Βρίσκεται στην ίδια στήλη αλλά γραμμές πιο κάτω.
					    	 //cuPrintf("at2 i = %d %d\n",at2,i);
					    	 
					    	 //κάνε xor 
					    	 in_gpu_data[at2]= in_gpu_data[at2] ^ in_gpu_data[at1];	 
					    	 
					     } // τελος if
				  

				  	  } // τελος for
			  
			  	  }//τελος if
			  
			  
			  __syncthreads(); 
			
}




/*
 * Η Συνάρτηση αυτή εκτελείται από τη GPU και βρίσκει τα μηδενικά που βρίσκονται κάτω από τη διαγώνιο.
 * Δέχεται σαν παραμέτρους τον πίνακα μετά από κάθε "κύκλο" του Gauss, τον αριθμό των στηλών, των γραμμών
 * και έναν πίνακα (gpu_lines_zeros) στον οποίο θα αποθηκευτεί ο αριθμός των μηδενικών κάθε στήλης.
 * Κάθε thread είναι υπεύθυνο να βρει τον αριθμό των μηδενικών της στήλης που είναι υπεύθυνα και να
 * αποθηκεύσει τον αριθμό αυτό στην αντίστοιχη θέση στον πίνακα gpu_lines_zeros πχ τα μηδενικά της
 * στήλης 0 θα αποθηκευτούν στη gpu_lines_zeros[0].
 * 
 */

__global__ void find_zeros(int *in_gpu_data, int cols, int lines, int *gpu_lines_zeros)
{

         int i = blockDim.x * blockIdx.x + threadIdx.x; 
	    //printf(" i = %d \n",i);
		//printf(" blockDim.x = %d \n",blockDim.x);
		//printf(" blockIdx.x = %d \n",blockIdx.x );
		//printf(" threadIdx.x = %d \n",threadIdx.x);
     
       if (i==1)
        {
           cuPrintf ( " ZEROS\n ");
        }
    
	      // printf("size = %d\n", size);
		   __syncthreads(); // Για να ξυπνήσουν όλα τα threads 
		
	
		   
		   if(i>=0 && i<cols-2)
		     {
			   int jump = cols+1;
			  // cuPrintf ("jump= %d\n",jump);
			   
			   // Η αρχική μου θέση είναι η πρώτη στήλη
			   // Κάθε thread αντιστοιχίζεται με τη στήλη που έχει τον ίδιο αριθμό με το global ID τους.
			   // Τα threads απλώνονται μέχρι και την προ-τελευταία στήλη (χωρίς αυτήν)
			  
			   int at1 = jump*i; //Θέση πάνω στη διαγώνιο για κάθε στήλη
			  // cuPrintf("start at1 = %d \n",at1);
			   
			   int at2 = at1 + cols; // Ο έλεγχος θα ξεκινήσει από τη θέση πάτω από τη διαγώνιο
			   //cuPrintf("katw thesh at2 = %d \n",at2);
			   
			   for(int t=i+1; t<lines; t++) // για να φτάσει μέχρι και την προ-προ τελευταία στήλη
			   {
				   if(in_gpu_data[at2] == 0) // αν έχει 0 τότε μέτρα το
				   {
					   gpu_lines_zeros[i]= gpu_lines_zeros[i]+1; // τα μηδενικά σε κάθε στήλη
				   }
				   
				   at2 = at2+cols;//κατέβα στην ακριβώς από κάτω θέση (ίδια στήλη επόμενη γραμμή)
				   
			   }
			   		   		   
		   }
		   
		   
		   
		   __syncthreads(); 
		  
			  
}




////////////////////////////////////////////////////////////////////////////////
//////////////////////////// MAIN /////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////


	 
int main(int argc , char *argv[])
{
	
	printf ("The number of arguments are %d \n", argc);
	
	for(int i=0;i<argc;i++)
	  {
		printf("%s\n",argv[i]);
	  }
		
		
	 
  
	 // data is a 1D array where the gauss array will be saved, already allocated into the host, it's been allocated
     in_data = ReadFile(argv[1]); 
	
     /*
     for(int u=0; u<info.Size;u++)
     {
    	 printf("d[%d] = %d \n",u,in_data[u]);
     }
     */
	printf("number of lines %d \n", info.lines);
	printf("number of cols %d \n", info.cols);
	printf("size %d \n", info.Size);
	
		
	printf("END \n");
	
	
	
	
	
	 /*
        //start = clock();
        hipEvent_t time1,time2;
        hipEventCreate(&time1);
        hipEventCreate(&time2);
	
        hipEventRecord(time1);
	 */

	
	      //Πίνακας που θα αποθηκεύει 1 σε όποια γραμμή έχει 1 στην ίδια στήλη με το pivot. Aρχικοποίηση με 0.
		  lines_one = (int*)malloc(sizeof(int*)*info.lines);
			
	      for(int h=0; h< info.lines ; h++)
	      {
	         lines_one[h]=0;
	      }
	      printf("END 6\n");
	 
	      
	      //Πίνακας που θα αποθηκεύει τον αριθμό των μηδενικών κάθε στήλης 
	      // στην αντίστοιχη θέση του. Aρχικοποίηση με 0.
	      lines_zeros = (int*)malloc(sizeof(int*)*info.lines);
 
	      for(int h2=0; h2< info.lines ; h2++)
	      	      {
	      	         lines_zeros[h2]=0;
	      	      }
	      printf("END 6.1\n");
    
	      
	      
    int h_start_col = 0; // αρχικοποίηση της στήλης που θα βρισκόμαστε κάθε φορά
	int h_start_line = 0; // αρχικοποίηση της γραμμής που θα βρισκόμαστε κάθε φορά
	int at = 0; // αρχικη θεση που ξεκιναμε για γραμμή και στήλη 0
	//int h_anw = 0;
	
	//Εύρεση του αριθμού των μηδενικών που πρέπει να υπάρχουν κάτω από τη διαγώνιο για να θεωρείται ο πίνακας άνω τριγωνικός
	int Anw_Trigwnikos = ((info.lines * info.lines)-info.lines)/2;
	printf( "number of zeros = %d \n", Anw_Trigwnikos);
  
	

	int k=0;
  
	printf("END 1\n");
  
	//Κάθε επανάληψη είναι ένας κύκλος κατά την εφαρμογή του Gauss.
	while ( k==0)
	{
  
       
        printf(" start_col ///// 1 = %d\n", h_start_col);
		printf( "start_line ////// 1 = %d\n", h_start_line);
  
  
		//  hipMalloc
		// Allocate to the device memory for the array
	
		gpuErrchk( hipMalloc((void **)&in_gpu_data, sizeof(int*)*info.Size));

		printf("END 2\n");
		
         
	
		// hipMemcpy
	
		// Copy the host input (in_data) in host memory to the device input (in_gpu_data) device memory 
    
		printf("Copy input data from the host memory to the CUDA device\n");
	
		gpuErrchk(hipMemcpy(in_gpu_data, in_data, sizeof(int*)*info.Size, hipMemcpyHostToDevice));

		printf("END 3\n");
	
			
         
		 // at  η θεση στην οποια βρισκομστε κάθε φορά
			
		if(h_start_col ==  h_start_line) // αν η γραμμή στην οποία βρισκόμαστε ταυτιζεται με τη στήλη στην οποία βρισκόμαστε
			{
				if(h_start_col != 0) // και δεν είναι η γραμμή και στήλη 0
				{
					at = at + info.cols + 1; // τότε η επόμενη θέση pivot είναι η επόμεν θέση πάνω στη διαγώνιο
				}
			}
		else if (h_start_col > h_start_line) // αλλιώς αν ο αριθμός της στήλης που βρισκόμαστε είναι μεγαλύτερος από αυτόν της γραμμής
			{
				at = at + info.cols + 1; // τότε η επόμενη θέση είναι στην επόμενη γραμμή και επόμενη στήλη από αυτές στις οποίες βρισκόμαστε
			}
		
		//printf (".///// at = %d\n", at);
		//printf (" [%d,%d] = %d\n",h_start_line,h_start_col,in_data[at]);
	  
/**********************************************************************************************************************************/	  
	  
	  
	    printf("END 5\n");
	 
		// αν η θεση στην οποια βρισκομστε ειναι 0 θα πρέπει να βρούμε μια γραμμή που έχει 1 στην ίδια στήλη στην οποία βρισκόμαστε
		if ( in_data[at] == 0)
			{
	           	printf("END 5.1\n");
				
				int swap_line=0; // η γραμμή με την οποία θα κάνει ανταλλαγή / αρχικοποίηση με 0 
	  
				while ( swap_line == 0 )
					{
	  
	                    printf("END 5.2\n");
								
						// ευρεση της γραμμς που θα γινει η ανταλλαγή
	                    // αν δε βρει στην ίδια στήλη κάποια γραμμή με 0 τοτε θα συνεχίσει να ψάχνει στην ακριβώς επόμενη στήλη
	                    
	                      for( int i = 0; i < (info.lines-h_start_line); i++ ) 
							{
	                    	  
	                    	   int at2 = at + (info.cols * i) ; // οι θέσεις κάτω ακριβώς από τη pivot
		                       // printf(" t2 -= %d\n", at2);
							   if ( in_data[at2] == 1 ) //αν είναι 1 τότε κράτα τη γραμμή
									{
								        //printf(" t23 -= %d\n", at2);
										swap_line = h_start_line+i;
										break; //for
									}
	  
	  
							}
		 
		                  	printf("END 5.3\n");
		                  	
						  if ( swap_line == 0 ) // δηλαδή δε βρήκε γραμμή για να κάνει swap
							 {
							    printf("END 5.4\n");
		      
							    h_start_col++; //αλλαγή στήλης
							    at++; // πηγαίνει στη δίπλα θέση
							}
						  else
							{		
								break;//while
							}
							
						
					} // τελος while
	
	
				printf("END 5.4\n");
		 			 
				printf(" swp line = %d\n", swap_line);
				
				printf(" SWAP\n");
				cudaPrintfInit();
				// καλώ τον kernel swap_ για να κάνει την ανταλλαγή γραμμών παράλληλα
				swap<<< 47, 64 >>>(in_gpu_data, info.cols, info.lines, at, h_start_col, h_start_line,swap_line); 
				gpuErrchk(hipGetLastError());
 
	 
	            printf("END 5.5\n");
	 
				gpuErrchk(hipMemcpy(in_data, in_gpu_data, sizeof(int*)*info.Size, hipMemcpyDeviceToHost));
				//^ σταματαει τη cpu και περιμενει τον kernel να τελειωσει για να παρει αποτελεσματα
				cudaPrintfDisplay(stdout,true);
				cudaPrintfEnd();
				gpuErrchk(hipDeviceReset());  // αδειάζει τη gpu τελείως


                printf("END 5.6\n");

                //Επομένως πρέπει να ξανακάνει allocate ότι χρειάζεται
				// hipMalloc
	
				gpuErrchk( hipMalloc((void **)&in_gpu_data, sizeof(int*)*info.Size));

				printf("END 5.7\n");
		

				// hipMemcpy
	
				// Copy the host input (in_data) in host memory to the device input (in_gpu_data)
				// device memory
    
				printf("Copy input data from the host memory to the CUDA device\n");
	
				gpuErrchk(hipMemcpy(in_gpu_data, in_data, sizeof(int*)*info.Size, hipMemcpyHostToDevice));

				printf("END 5.9\n");
	
			}	
			
		

	   
	  /*	 for(int u=0; u<info.Size;u++)
		     {
		    	 printf("d[%d] 2 = %d \n",u,in_data[u]);
		     }

	  */ 
	   
		
	   printf("END 6\n");
    /******************************************************************************************************************************/
		// XOR 
	   
	   printf("END 6.1\n");
	   //allocate memory for gpu_lines_one
	   gpuErrchk( hipMalloc((void **)&gpu_lines_one, sizeof(int*)*info.lines));
	   
	   //Ψάχνει να βρεις ποιες γραμμές έχουν 1 στην ίδια στήλη με το pivot, κάτω από τη γραμμή του pivot.
	   //Όποια έχει 1 τότε θα μπει 1 στην αντίστοιχη θέση του πίνακα gpu_lines_one
	   //Αυτές οι γραμμές για γίνουν xor με τη γραμμή στην οποία ανήκει το pivot
	   
	   int perisema = info.lines-h_start_line;
	   //printf(" perisems = %d \n", perisema);
	  
	   int c_l=h_start_line;
	   //printf("c_l 1= %d \n",c_l);
	   
	   for(int f=1; f < perisema; f++)
	   {
		   //printf("f = %d \n", f);
		   c_l++;
		   int x= at+info.cols*f;// η θέση η οποία ελέγχεται κάτω από το pivot
		   //printf("x = %d \n",x);
		   
		   if (in_data[x]==1) 
		   {
			   //printf("ffffffffffffff \n");
			   //printf("cl = %d \n",c_l);
			   lines_one[c_l] = 1;
			  	   
		   }

	   }
	   
	 /*   for(int g=0;g<info.lines;g++)
	  	   { 
	  		   printf(" g  hF = %d %d\n",g,lines_one[g]);
	  		   
	  	   }
	  */ 
	  
	   printf("END 6.2\n");
	   //στειλε τον πίνακα στη gpu
	   gpuErrchk( hipMemcpy(gpu_lines_one, lines_one, sizeof(int*)*info.lines, hipMemcpyHostToDevice));

	   printf(" XOR\n");
	   cudaPrintfInit();
	   // Καλώ τον kernel xor για να κάνει xor όσες γραμμές έχουν 1 στη στήλη του pivot
		xor_<<<47, 64 >>>(in_gpu_data, info.cols, info.lines, at,h_start_col,h_start_line,gpu_lines_one);
		gpuErrchk(hipGetLastError());

        printf("END 6.3\n");
      
        gpuErrchk(hipMemcpy(in_data, in_gpu_data, sizeof(int*)*info.Size, hipMemcpyDeviceToHost));
        //^ σταματαει τη cpu και περιμενει τον kernel να τελειωσει για να παρει αποτελεσματα
        cudaPrintfDisplay(stdout,true);
        cudaPrintfEnd();
		
		printf("END 6.4\n");
		
	/*	 for(int u=0; u<info.Size;u++)
		     {
		    	 printf("d[%d] 3 = %d \n",u,in_data[u]);
		     }
*/
		
	/********************************************************************************************************************************************************************/
		
	    /////// Ελεγχος για το αν ο πινακας εγινε ανω τριγωικός
           
		   //allocate memory to gpu, send the array as well (αρχικοποιημένο με 0)
		gpuErrchk( hipMalloc((void **)&gpu_lines_zeros, sizeof(int*)*info.lines));
		gpuErrchk( hipMemcpy(gpu_lines_zeros, lines_zeros, sizeof(int*)*info.lines, hipMemcpyHostToDevice));
		  
		printf(" ZEROS\n");
		cudaPrintfInit();
		// καλώ τον kernel find_zeros για να βρει τα μηδενικά που έχει κάθε στήλη κάτω από τη διαγώνιο
		find_zeros<<<47, 64 >>>(in_gpu_data, info.cols, info.lines, gpu_lines_zeros);
		gpuErrchk(hipGetLastError());
			  			
			  	    
		printf("END 6.3\n");
			  	      
		//παίρνω τον πίνακα με τα μηδενικά
		gpuErrchk(hipMemcpy(lines_zeros, gpu_lines_zeros, sizeof(int*)*info.lines, hipMemcpyDeviceToHost));
		 //^ σταματαει τη cpu και περιμενει τον kernel να τελειωσει για να παρει αποτελεσματα
		cudaPrintfDisplay(stdout,true);
		cudaPrintfEnd();
			  			
		printf("END 6.4\n");
               
		   
		int zeros=0;
		//μετράω πόσα είναι όλα τα μηδενικά
		for(int p=0;p<info.lines;p++)
			  {
				  zeros = zeros + lines_zeros[p];
				  
			  }
			
         
		// αν ο αριθμός των μηδενικών είναι ίσος με αυτόν που θέλουμε για να γίνει άνω τριγωνικός στμάτα τη while 
           if (zeros == Anw_Trigwnikos)
           {
        	   k=1;
           }
		
        printf("END 7\n");
        
        //αλλαγή γραμμής και στήλης σε περίπτωση που δεν είναι ανω τριγωνικός
        h_start_col++;
        h_start_line++;
        		
        printf(" start_col ///// 2= %d\n", h_start_col);
        printf( "start_line ////// 2= %d\n", h_start_line);
        		
        		
        printf("END 6.3\n");
        
    	
    	gpuErrchk(hipDeviceReset());     

    	 for(int g=0;g<info.lines;g++)
    		   {
    			  //ξαναμηδενίζουμε τους πίνακες για να ξαναχρησιμοποιηθούν
    			  // printf(" g  hF = %d %d\n",g,lines_one[g]);
    			  lines_one[g]=0;
    			  lines_zeros[g]=0;
    			   
    		   }
    
    
  }
    
    printf("END 7.1\n");
    
   
    gpuErrchk(hipPeekAtLastError());
   
	/*
	for(int i = 0; i < info.Size; i++)
    {
        printf("data[%d]=%d\n", i, out_data[i]);
    }
	printf("END 10 \n");
	
//////////////////////////////


hipEventRecord(time2);

printf("END 10.1 \n");
    
    hipEventSynchronize(time2);
    printf("END 10.2 \n");
    float totalTime =0;
    printf("END 10.3 \n");
    hipEventElapsedTime(&totalTime, time1, time2);
    printf("END 10.4 \n");
	
	
*/

printf("END 10.4 \n");

WriteFile(in_data);

printf("END 10.5 \n");




    
    
   
    // Free host memory
    free(in_data);
    free(lines_one);
    free(lines_zeros);
    hipProfilerStop();
    printf("END 11 \n");

   printf("Done\n");
  //printf("Total time elapsed = %5.2f ms\n", totalTime);
  // printf("Total time elapsed = %f \n", totalTime);
     

	 
	 
    return 0;

}// end of main


	 


	 
